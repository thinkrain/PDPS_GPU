#include "hip/hip_runtime.h"
/* ----------------------------------------------------------------------
PDPS - Particle Dynamics Parallel Simulator

Copyright (2012) reserved by Lingqi Yang.
Email: ly2282@columbia.edu

See the README file in the top-level PDPS directory.
------------------------------------------------------------------------- */

#include "math.h"
#include "stdio.h"
#include "stdlib.h"
#include "string.h"
#include "parallel.h"
#include "error.h"
#include "domain.h"
#include "force.h"
#include "memory.h"
#include "neighbor.h"
#include "neigh_list.h"
#include "pair_sph_rhosumlocalAV.h"
#include "particle.h"
#include "random_mars.h"
#include "update.h"
#include "group.h"

#include "pdps_cuda.h"
#include "cuda_engine.h"
#include ""
#include "hip/device_functions.h"
using namespace PDPS_NS;

#define DELTA 1
#define EPSILON 1.0e-10
#define PI 3.1416

__global__ void gpuComputerhoAV(double *devCoordX, double *devCoordY, double *devCoordZ, int *devPairtable, int *devNumneigh,
					double *devRho, double *devVolume, double *devPoro, double *devMass, int *devType, int *devMask, const double h,
					const int nlocal, const double a3D, const int lgroupbit, const int sgroupbit, int *devSetflag, double *devCutsq, 
					const int inclusion_flag, double *devRho0){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	double wf, xtemp, ytemp, ztemp, rsq, delx, dely, delz, q, imass, jmass;
	int j, jj, itype, jtype, jnum;
	__shared__ double mass[TYPEMAX];
	__shared__ double rho0[TYPEMAX];
	__shared__ int setflag[TYPEMAX * TYPEMAX];
	__shared__ double cutsq[TYPEMAX * TYPEMAX];

	for (int tid = 0; tid < TYPEMAX; tid++){
		mass[tid] = devMass[tid];
		rho0[tid] = devRho0[tid];
		for (j = 0; j < TYPEMAX; j++){
			setflag[tid * TYPEMAX + j] = devSetflag[tid * TYPEMAX + j];
			cutsq[tid * TYPEMAX + j] = devCutsq[tid * TYPEMAX + j];
		}
			
	}

	for (i = i; i < nlocal; i += blockDim.x * gridDim.x){
		itype = devType[i];
		wf = a3D;
		if (inclusion_flag == 2){
			if (devMask[i] & lgroupbit){
				imass = mass[itype];
				devRho[i] = imass * wf;
				devPoro[i] = 0.0;

			}
			if (devMask[i] & sgroupbit){
				devPoro[i] = devRho[i] / rho0[itype];
			}
		}
		else{
			if (devMask[i] & lgroupbit){
				imass = mass[itype];
				devRho[i] = imass * wf;
				devPoro[i] = 0.0;
			}
			else if (devMask[i] & sgroupbit)
				devPoro[i] = devVolume[i] * wf;
		}

	}
	__syncthreads();
	for (i = blockIdx.x * blockDim.x + threadIdx.x; i < nlocal; i += blockDim.x * gridDim.x){
		xtemp = devCoordX[i];
		ytemp = devCoordY[i];
		ztemp = devCoordZ[i];
		jnum = devNumneigh[i];
		for (jj = 0; jj < jnum; jj++){
			j = devPairtable[i * NEIGHMAX + jj];
			jtype = devType[j];
			if (setflag[itype * TYPEMAX + jtype]){
				delx = xtemp - devCoordX[j];
				dely = ytemp - devCoordY[j];
				delz = ztemp - devCoordZ[j];
				rsq = delx * delx + dely * dely + delz * delz;
				if (rsq < cutsq[itype * TYPEMAX + jtype]){
					q = sqrt(rsq) / h;
					//	Cubic Spline
					if (q < 1)
						wf = 1 - 1.5 * q * q + 0.75 * q * q * q;
					else
						wf = 0.25 * (2 - q) * (2 - q) * (2 - q);
					wf = wf * a3D;
					if (inclusion_flag == 2){
						if (devMask[i] & lgroupbit){
							devRho[i] += mass[jtype] * wf;
						}
						//if (devMask[j] & lgroupbit){
						//	devRho[j] += mass[itype] * wf;
						//}
						//if (devMask[i] & sgroupbit)
						//	devPoro[j] += mass[itype] * wf / rho0[itype];
						if (devMask[j] & sgroupbit)
							devPoro[i] += mass[jtype] * wf / rho0[jtype];
					}
					else{
						//  detect solid particle for local average
						if (devMask[i] & sgroupbit && devMask[j] & lgroupbit){
							devPoro[j] += devVolume[i] * wf;
							if (inclusion_flag == 1)
								devRho[j] += rho0[jtype] * devVolume[i] * wf;

						}
						else if (devMask[j] & sgroupbit && devMask[i] & lgroupbit){
							devPoro[i] += devVolume[j] * wf;
							if (inclusion_flag == 1)
								devRho[i] += rho0[itype] * devVolume[j] * wf;
						}
						else if (devMask[j] & sgroupbit && devMask[i] & sgroupbit){
							devPoro[j] += devVolume[i] * wf;
							devPoro[i] += devVolume[j] * wf;
						}
						else {
							if (devMask[i] & lgroupbit){
								devRho[i] += mass[jtype] * wf;
							}

							if (devMask[j] & lgroupbit){
								devRho[j] += mass[itype] * wf;
							}
						}
					}
					////  detect solid particle for local average
					//if (devMask[j] & sgroupbit){
					//	devPoro[i] += devVolume[j] * wf;
					//}
					//else if (devMask[i] & sgroupbit){
					//	devPoro[i] += devVolume[i] * wf;
					//}
					//else {
					//	if (devMask[i] & lgroupbit){
					//		devRho[i] += mass[jtype] * wf;
					//	}
					//}

				}		//  rsq < cutsq[itype][jtype]

			}	// setflag[itype * 10 + jtype]
		}	// j < jnum

	}	// i < nlocal
	if (inclusion_flag == 0){
		if (i < nlocal){
			if (devMask[i] & lgroupbit)
				devRho[i] = devRho[i] / (1 - devPoro[i]);
		}
	}

}


/* ---------------------------------------------------------------------- */

PairSPH_RHOSUMLOCALAV::PairSPH_RHOSUMLOCALAV(PDPS *ps) : Pair(ps)
{
	first = 1;
	newton_pair = 1;
	allocated = 0;
	cubic_flag = 0;
	quintic_flag = 0;
	poro_flag = 0;
	inclusion_flag = 0;
	h = 0.0;
	cut = NULL;
	cutsq = NULL;
	comm_forward = comm_reverse = 2;
	devSetflag = NULL;
	devCutsq = NULL;

}

/* ---------------------------------------------------------------------- */

PairSPH_RHOSUMLOCALAV::~PairSPH_RHOSUMLOCALAV()
{
	if (allocated) {
		memory->destroy(setflag);
		memory->destroy(cutsq);
		memory->destroy(rho0);
		memory->destroy(cut);

	}

}

/* ---------------------------------------------------------------------- */

void PairSPH_RHOSUMLOCALAV::allocate()
{
	allocated = 1;
	int n = particle->ntypes;

	memory->create(setflag, n + 1, n + 1, "pair:setflag");
	for (int i = 1; i <= n; i++)
	for (int j = i; j <= n; j++)
		setflag[i][j] = 0;
	memory->create(cutsq, n + 1, n + 1, "pair:cutsq");
	memory->create(rho0, n + 1, "pair:rho0");
	memory->create(cut, n + 1, n + 1, "pair:cut");

	// pointer to transfer to GPU
	hostSetflag = (int *)malloc(TYPEMAX * TYPEMAX * sizeof(int));
	hostCutsq = (double *)malloc(TYPEMAX * TYPEMAX * sizeof(double));
}

/* ----------------------------------------------------------------------
Compute force for all paritcles
------------------------------------------------------------------------- */

void PairSPH_RHOSUMLOCALAV::compute(int eflag, int vflag)
{
	int i, j, ii, jj, inum, jnum, itype, jtype;
	double xtmp, ytmp, ztmp, delx, dely, delz, fpair;

	int *ilist, *jlist, *numneigh, **firstneigh;
	double vxtmp, vytmp, vztmp, imass, jmass, fi, fj, fvisc, q;
	double rsq, rij_inv, tmp, wfd, delVdotDelR, mu, deltaE;
	MPI_Request request;
	MPI_Status status;
	if (eflag || vflag)
		ev_setup(eflag, vflag);
	//  else
	//    evflag = vflag_fdotr = 0;

	double **v = particle->vest;
	double **x = particle->x;
	double **f = particle->f;
	double *rho = particle->rho;
	double *mass = particle->mass;
	double *de = particle->de;
	double *drho = particle->drho;
	double *poro = particle->poro;
	double *volume = particle->volume;
	int *type = particle->type;
	int *mask = particle->mask;
	int nlocal = particle->nlocal;
	double wf;
	//  int newton_pair = force->newton_pair;
	// check consistency of pair coefficients
	if (first) {
		for (i = 1; i <= particle->ntypes; i++) {
			for (j = 1; i <= particle->ntypes; i++) {
				if (cutsq[i][j] > 1.e-32) {
					if (!setflag[i][i] || !setflag[j][j]) {
						if (parallel->procid == 0) {
							printf(
								"SPH particle types %d and %d interact with cutoff=%g, but not all of their single particle properties are set.\n",
								i, j, sqrt(cutsq[i][j]));
						}
					}
				}
			}
		}
		first = 0;
	}

	inum = neighbor->neighlist->inum;
	ilist = neighbor->neighlist->ilist;
	numneigh = neighbor->neighlist->numneigh;
	firstneigh = neighbor->neighlist->firstneigh;

	// loop over neighbors of my particles
//	printf("before pair timestep = %d procid = %d rho[0] = %f\n", update->ntimestep, parallel->procid, rho[0]);
	//if ((update->ntimestep % nstep) == 0) {

	//	// initialize density with self-contribution,
	//	for (i = 0; i < nlocal; i++) {
	//		itype = type[i];
	//		if (domain->dim == 3) {

	//			// Cubic spline kernel, 3d
	//			wf = a3D;
	//		}
	//		else {
	//			// Cubic spline kernel, 2d
	//			wf = a2D;
	//		}
	//		if (mask[i] & lgroupbit){
	//			imass = mass[itype];
	//			rho[i] = imass * wf;
	//			poro[i] = 0.0;
	//		}
	//		else if (mask[i] & sgroupbit){
	//			poro[i] = volume[i] *wf;
	//		}

	//		
	//	}
	//	//	set all ghost particle's rho and porosity zero to be computed
	//	if (particle->nghost > 0){
	//		for (i = nlocal; i < nlocal + particle->nghost; i++){
	//			itype = type[i];
	//			if (mask[i] & lgroupbit){
	//				rho[i] = 0.0;
	//				poro[i] = 0.0;
	//			}
	//			if (mask[i] & sgroupbit){
	//				poro[i] = 0.0;
	//			}
	//				
	//		}
	//	}
	//	for (ii = 0; ii < inum; ii++) {
	//		i = ilist[ii];
	//		xtmp = x[i][0];
	//		ytmp = x[i][1];
	//		ztmp = x[i][2];
	//		itype = type[i];
	//		jlist = firstneigh[i];
	//		jnum = numneigh[i];

	//		for (jj = 0; jj < jnum; jj++) {
	//			j = jlist[jj];

	//			jtype = type[j];
	//			if (setflag[itype][jtype]){
	//				delx = xtmp - x[j][0];
	//				dely = ytmp - x[j][1];
	//				delz = ztmp - x[j][2];
	//				rsq = delx * delx + dely * dely + delz * delz;

	//				if (rsq < cutsq[itype][jtype]) {
	//					q = sqrt(rsq) / h;

	//					if (cubic_flag == 1){
	//						if (q < 1)
	//							wf = 1 - 1.5 * q * q + 0.75 * q * q * q;
	//						else
	//							wf = 0.25 * (2 - q) * (2 - q) * (2 - q);
	//					}
	//					else if (quintic_flag == 1)
	//						wf = (1 - q / 2.0) * (1 - q / 2.0) * (1 - q / 2.0) * (1 - q / 2.0) * (2 * q + 1);

	//					if (domain->dim == 3)
	//						wf = wf * a3D;
	//					else
	//						wf = wf * a2D;

	//					//  detect solid particle for local average
	//					if (mask[i] & sgroupbit && mask[j] & lgroupbit){
	//						poro[i] += volume[i] * wf;
	//						poro[j] += volume[i] * wf;
	//						if (inclusion_flag == 1)
	//							rho[j] += rho0[jtype] * volume[i] * wf;

	//					}
	//					else if (mask[j] & sgroupbit && mask[i] & lgroupbit){
	//						poro[j] += volume[j] * wf;
	//						poro[i] += volume[j] * wf;
	//						if (inclusion_flag == 1)
	//							rho[i] += rho0[itype] * volume[j] * wf;
	//					}
	//					else if (mask[j] & sgroupbit && mask[i] & sgroupbit){
	//						poro[j] += volume[i] * wf;
	//						poro[i] += volume[j] * wf;
	//					}
	//					else {
	//						if (mask[i] & lgroupbit){
	//							rho[i] += mass[jtype] * wf;
	//						}

	//						if (mask[j] & lgroupbit){
	//							rho[j] += mass[itype] * wf;
	//						}
	//					}

	//				}

	//			}

	//		}

	//	}
	//
	//	parallel->reverse_comm_pair(this); 
	//	if (poro_flag == 1){
	//		for (i = 0; i < nlocal; i++){
	//			itype = type[i];
	//			if (mask[i] & lgroupbit)
	//				rho[i] = rho[i] / (1 - poro[i]);
	//		}
	//	}
	//	parallel->forward_comm_pair(this);


	//}
	//hipError_t error_t;
	//error_t = hipMemcpy(neighbor->hostForceX, particle->devForceX, particle->nlocal * sizeof(double), hipMemcpyDeviceToHost);
	
	gpuComputerhoAV << < GRID_SIZE, BLOCK_SIZE >> >(particle->devCoordX, particle->devCoordY, particle->devCoordZ,
		neighbor->devPairtable, neighbor->devNumneigh, particle->devRho, particle->devVolume, particle->devPoro, particle->devMass, 
		particle->devType, particle->devMask, h, nlocal, a3D, lgroupbit, sgroupbit, devSetflag, devCutsq, inclusion_flag, devRho0);
	//error_t = hipMemcpy(rho0, devRho0, TYPEMAX * sizeof(double), hipMemcpyDeviceToHost);
	//error_t = hipMemcpy(neighbor->hostForceX, particle->devForceX, particle->nlocal * sizeof(double), hipMemcpyDeviceToHost);
	//error_t = hipMemcpy(hostSetflag, devSetflag, TYPEMAX * TYPEMAX * sizeof(int), hipMemcpyDeviceToHost);
	//error_t = hipMemcpy(hostCutsq, devCutsq, TYPEMAX * TYPEMAX * sizeof(double), hipMemcpyDeviceToHost);
	//error_t = hipMemcpy(particle->ptrHostRho, particle->devRho, particle->nlocal * sizeof(double), hipMemcpyDeviceToHost);
	//error_t = hipMemcpy(particle->ptrHostPoro, particle->devPoro, particle->nlocal * sizeof(double), hipMemcpyDeviceToHost);
	//error_t = hipMemcpy(neighbor->hostNumneigh, neighbor->devNumneigh, particle->nlocal * sizeof(int), hipMemcpyDeviceToHost);
	//error_t = hipMemcpy(neighbor->hostPairtable, neighbor->devPairtable, particle->nlocal * NEIGHMAX * sizeof(int), hipMemcpyDeviceToHost);
	//error_t = hipMemcpy(hostSetflag, devSetflag, TYPEMAX * TYPEMAX * sizeof(int), hipMemcpyDeviceToHost);
	//error_t = hipMemcpy(hostCutsq, devCutsq, TYPEMAX * TYPEMAX * sizeof(double), hipMemcpyDeviceToHost);
}



/* ----------------------------------------------------------------------
Setting for pair_style command
------------------------------------------------------------------------- */

void PairSPH_RHOSUMLOCALAV::set_style(int narg, char **arg)
{
	//	if (narg != 4)
	//		error->all(FLERR, "Illegal number of setting arguments for pair_style sph/idealgas");
	if (strcmp(arg[1], "Cubic") == 0)
		cubic_flag = 1;
	else if (strcmp(arg[1], "Quintic") == 0)
		quintic_flag = 1;
	else
		error->all(FLERR, "Wrong Kernel function");
	nstep = atoi(arg[2]);

	lgid = group->find_group(arg[3]);
	if (lgid == -1) {
		char str[128];
		sprintf(str, "Cannot find group id: %s", arg[3]);
		error->all(FLERR, str);
	}
	lgroupbit = group->bitmask[lgid];

	sgid = group->find_group(arg[4]);
	if (sgid == -1) {
		char str[128];
		sprintf(str, "Cannot find group id: %s", arg[4]);
		error->all(FLERR, str);
	}
	sgroupbit = group->bitmask[sgid];

	if (strcmp(arg[5], "poro") == 0)
		poro_flag = 1;
	else if (strcmp(arg[5], "inclusion") == 0)
		inclusion_flag = 1;
	else if (strcmp(arg[5], "inclusion2") == 0)		//	record the porosity of one liquid
		inclusion_flag = 2;

}

/* ----------------------------------------------------------------------
Set Coeff for pair_coeff command
------------------------------------------------------------------------- */

void PairSPH_RHOSUMLOCALAV::set_coeff(int narg, char **arg)
{
	if (narg != 4)
		error->all(FLERR, "Incorrect args for pair_style sph/taitwater coefficients");
	if (!allocated)
		allocate();

	int ilo, ihi, jlo, jhi;

	force->bounds(arg[0], particle->ntypes, ilo, ihi);
	force->bounds(arg[1], particle->ntypes, jlo, jhi);

	double rho0_one = atof(arg[2]);
	double cut_one = atof(arg[3]);

	int count = 0;
	for (int i = ilo; i <= ihi; i++) {
		rho0[i] = rho0_one;
		for (int j = MAX(jlo, i); j <= jhi; j++) {
			rho0[j] = rho0_one;
			cut[i][j] = 2 * cut_one;
			cutsq[i][j] = cut[i][j] * cut[i][j];
			setflag[i][j] = 1;
			count++;
			hostSetflag[i * TYPEMAX + j] = setflag[i][j];
			hostSetflag[j * TYPEMAX + i] = hostSetflag[i * TYPEMAX + j];
			hostCutsq[i * TYPEMAX + j] = cutsq[i][j];
			hostCutsq[j * TYPEMAX + i] = hostCutsq[i * TYPEMAX + j];
		}
	}

	h = cut_one;

	if (cubic_flag == 1){
		a2D = 10.0 / 7.0 / PI / h / h;
		a3D = 1.0 / PI / h / h / h;
	}
	else if (quintic_flag == 1){
		a2D = 7.0 / 4.0 / PI / h / h;
		a3D = 21.0 / 16.0 / PI / h / h / h;
	}

	if (count == 0)
		error->all(FLERR, "Incorrect args for pair coefficients");

	// setup for GPU parameters
	hipError_t cudaStatus;
	hipMalloc(&devSetflag, TYPEMAX * TYPEMAX * sizeof(int));
	hipMalloc(&devCutsq, TYPEMAX * TYPEMAX * sizeof(double));
	hipMalloc(&devRho0, TYPEMAX * sizeof(double));
	hipMemcpy(devSetflag, hostSetflag, TYPEMAX * TYPEMAX * sizeof(int), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(devCutsq, hostCutsq, TYPEMAX * TYPEMAX * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(devRho0, rho0, TYPEMAX * sizeof(double), hipMemcpyHostToDevice);
}

/* ----------------------------------------------------------------------
init for one type pair i,j and corresponding j,i
------------------------------------------------------------------------- */

void PairSPH_RHOSUMLOCALAV::init_one(int i, int j) {

	if (setflag[i][j] == 0) {
		error->all(FLERR, "Not all pair sph/taitwater coeffs are set");
	}
	force->type2pair[i][j] = pair_id;

	cut[j][i] = cut[i][j];
	cutsq[j][i] = cutsq[i][j];
	setflag[j][i] = setflag[i][j];
	force->type2pair[j][i] = force->type2pair[i][j];


}

/* ---------------------------------------------------------------------- */

double PairSPH_RHOSUMLOCALAV::single(int i, int j, int itype, int jtype,
	double rsq, double factor_coul, double factor_lj, double &fforce) {
	fforce = 0.0;
	return 0.0;
}

/* ----------------------------------------------------------------------
pack particle's rho to neighbor processors during reverse communication
------------------------------------------------------------------------- */

int PairSPH_RHOSUMLOCALAV::pack_reverse_comm(int n, int first, double *buf) {
	int i, m, last;
	double *rho = particle->rho;
	double *poro = particle->poro;

	m = 0;
	last = first + n;
	for (i = first; i < last; i++) {
		buf[m++] = rho[i];
		buf[m++] = poro[i];
	}
	return m;
}

/* ----------------------------------------------------------------------
unpack particle's rho from neighbor processors during reverse communication
------------------------------------------------------------------------- */

void PairSPH_RHOSUMLOCALAV::unpack_reverse_comm(int n, int *list, double *buf) {
	int i, m, j;
	double *rho = particle->rho;
	int *mask = particle->mask;
	double *poro = particle->poro;
	int jtype;
	m = 0;

	for (i = 0; i < n; i++) {
		j = list[i];
		if (mask[j] & lgroupbit)
			rho[j] += buf[m++];
		else
			m++;
		poro[j] += buf[m++];
	}
}

/* ----------------------------------------------------------------------
pack particle's rho to neighbor processors during forward communication
------------------------------------------------------------------------- */

int PairSPH_RHOSUMLOCALAV::pack_forward_comm(int n, int *list, double *buf) {
	int i, m, j;
	double *rho = particle->rho;
	double *poro = particle->poro;

	m = 0;
	for (i = 0; i < n; i++) {
		j = list[i];
		buf[m++] = rho[j];
		buf[m++] = poro[j];
	}
	return m;
}

/* ----------------------------------------------------------------------
unpack particle's rho from neighbor processors during forward communication
------------------------------------------------------------------------- */

void PairSPH_RHOSUMLOCALAV::unpack_forward_comm(int n, int first, double *buf) {
	int i, m, last;
	double *rho = particle->rho;
	double *poro = particle->poro;

	m = 0;
	last = first + n;
	for (i = first; i < last; i++) {
		rho[i] = buf[m++];
		poro[i] = buf[m++];
	}

}