#include "hip/hip_runtime.h"
/* ----------------------------------------------------------------------
   PDPS - Particle Dynamics Parallel Simulator

   Copyright (2012) reserved by Lingqi Yang. 
   Email: ly2282@columbia.edu

   See the README file in the top-level PDPS directory.
------------------------------------------------------------------------- */

#include "math.h"
#include "stdio.h"
#include "stdlib.h"
#include "string.h"
#include "neighbor.h"
#include "create_particle.h"
#include "domain.h"
#include "error.h"
#include "group.h"
#include "memory.h"
#include "parallel.h"
#include "particle.h"
#include "particle_type.h"
#include "phy_const.h"
#include "random_park.h"
#include "style_particle.h"
#include "timer.h"

#include "pdps_cuda.h"
#include "cuda_engine.h"
#include ""
#include "hip/device_functions.h"
using namespace PDPS_NS;
using namespace PhyConst;

#define DELTA 10000
#define EPSILON 1.0e-6



__global__ void gputest(double *devCoordX){
	int pid = blockIdx.x * blockDim.x + threadIdx.x;
		devCoordX[pid] = -1;
}
//	convert from Aos to SoA
__global__ void gpuInterLeave(double *ArrayRaw, double *ArrayX, double *ArrayY, double * ArrayZ, const int nlocal){
	extern __shared__ double buf[];
	int pid = blockIdx.x * blockDim.x + threadIdx.x;
	if (pid < nlocal){
		buf[threadIdx.x * 3 + 0] = ArrayX[pid];
		buf[threadIdx.x * 3 + 1] = ArrayY[pid];
		buf[threadIdx.x * 3 + 2] = ArrayZ[pid];
	}
	__syncthreads();
	int g = blockIdx.x * blockDim.x * 3;
	int p = threadIdx.x;
	if (g + p < nlocal * 3)
		ArrayRaw[g + p] = buf[p];
	p += blockDim.x;
	if (g + p < nlocal * 3)
		ArrayRaw[g + p] = buf[p];
	p += blockDim.x;
	if (g + p < nlocal * 3)
		ArrayRaw[g + p] = buf[p];

}

//	convert from SoA to Aos
__global__ void gpuDeinterLeave(double *ArrayRaw, double *ArrayX, double *ArrayY, double * ArrayZ, const int nlocal){
	extern __shared__ double buf[];

	int g = blockIdx.x * blockDim.x * 3;
	int p = threadIdx.x;
	if (g + p < nlocal * 3)
		buf[p] = ArrayRaw[g + p];
	p += blockDim.x;
	if (g + p < nlocal * 3)
		buf[p] = ArrayRaw[g + p];
	p += blockDim.x;
	if (g + p < nlocal * 3)
		buf[p] = ArrayRaw[g + p];
	__syncthreads();

	int pid = blockIdx.x * blockDim.x + threadIdx.x;
	if (pid < nlocal){
		ArrayX[pid] = buf[threadIdx.x * 3 + 0];
		ArrayY[pid] = buf[threadIdx.x * 3 + 1];
		ArrayZ[pid] = buf[threadIdx.x * 3 + 2];
	}



}

// copy data between device
template<class TYPE> __global__ void gpuCopy(
	TYPE* __restrict Out,
	TYPE* __restrict In,
	const int  n)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n) Out[i] = In[i];
}


Particle::Particle(PDPS *ps) : Pointers(ps)
{


	x = NULL;
	v = NULL;
	f = NULL;
	tag = NULL;
	type = NULL;
	mass = NULL;
	mask = NULL;

	density = NULL;
	omega = NULL;
	radius = NULL;
	poro = NULL;
	volume = NULL;
	hlocal = NULL;
	rmass = NULL;
	torque = NULL;

	ptype = NULL;

	map_array = NULL;

	// Default value
	nparticles = 0;          // default number of particles
	nlocal = nghost = 0;
	nmax = 0;                // default number of materials
	nfirst = 0;
	ntypes = 0;              // default number of types
	//maxarg = 0;            // max argument to allocate memory

	vest = NULL;
	rho = NULL;
	drho = NULL;
	e = NULL;
	de = NULL;
	cv = NULL;

	// particle type flag
	atomic_flag = 1;
	sphere_flag = 0;
	rmass_flag = radius_flag = omega_flag = torque_flag = 0;
	ee_flag = rho_flag = cv_flag = vest_flag = 0;

	tag_enable = 1;
	map_style = 0;
	map_tag_max = 0;
	map_nhash = 0;

	// used by read_data class
	size_data_atom = 5;
	size_data_vel = 4;
	xcol_data = 3;

	nprimes = 38;
	primes = new int[nprimes];
	int plist[] = {5041,10007,20011,30011,40009,50021,60013,70001,80021,
				   90001,100003,110017,120011,130003,140009,150001,160001,
				   170003,180001,190027,200003,210011,220009,230003,240007,
				   250007,260003,270001,280001,290011,300007,310019,320009,
				   330017,340007,350003,362881,3628801};
	for (int i = 0; i < nprimes; i++) primes[i] = plist[i];

	particle_style = NULL;
	ptype = NULL;
	create_particle_type("atomic", 0, NULL);

	//	pointer to GPU device
	devCoordX = NULL;
	devCoordY = NULL;
	devCoordZ = NULL;
	devCoordXold = NULL;
	devCoordYold = NULL;
	devCoordZold = NULL;
	devVeloX = NULL;
	devVeloY = NULL;
	devVeloZ = NULL;
	devVestX = NULL;
	devVestY = NULL;
	devVestZ = NULL;
	devForceX = NULL;
	devForceY = NULL;
	devForceZ = NULL;
	devMask = NULL;
	devMass = NULL;
	devTag = NULL;
	devType = NULL;
	devRho = NULL;
	devRadius = NULL;
	devRmass = NULL;
	devDensity = NULL;
	devPoro = NULL;
	devVolume = NULL;

	devHostCoord = NULL;
	devHostVelo = NULL;
	devHostVest = NULL;
	devHostForce = NULL;
	devHostMask = NULL;
	devHostMassType = NULL;
	devHostTag = NULL;
	devHostType = NULL;
	devHostRho = NULL;
	devHostRadius = NULL;
	devHostRmass = NULL;
	devHostDensity = NULL;
	devHostPoro = NULL;
	devHostVolume = NULL;

	ptrHostCoord = NULL;
	ptrHostVelo = NULL;
	ptrHostVest = NULL;
	ptrHostForce = NULL;
	ptrHostMask = NULL;
	ptrHostMassType = NULL;
	ptrHostTag = NULL;
	ptrHostType = NULL;
	ptrHostRho = NULL;
	ptrHostRadius = NULL;
	ptrHostRmass = NULL;
	ptrHostDensity = NULL;
	ptrHostPoro = NULL;
	ptrHostVolume = NULL;
		

}

/* ---------------------------------------------------------------------- */

Particle::~Particle()
{
	memory->destroy(x);
	memory->destroy(v);
	memory->destroy(f);
	memory->destroy(type);
	memory->destroy(mask);

	delete[] mass;
	mass = NULL;
}

/* ---------------------------------------------------------------------- */

void Particle::init()
{
	if (nparticles == 0) {
		error->all(FLERR,"No particle has been created");
	}

  	ptype->init();
}

/* ---------------------------------------------------------------------- */

void Particle::create_particle_type(const char *style, int narg, char **arg)
{
	delete [] particle_style;
	if (ptype) delete ptype;

	if (0) return;

#define PARTICLE_CLASS
#define ParticleStyle(key,Class) \
	else if (strcmp(style,#key) == 0) ptype = new Class(ps,narg,arg);
#include "style_particle.h"
#undef ParticleStyle
#undef PARTICLE_CLASS

	else error->all(FLERR, "Invalid particle style");

	int n = strlen(style) + 1;
	particle_style = new char[n];
	strcpy(particle_style, style);
}



/* ----------------------------------------------------------------------
   Add tag for each created particle
------------------------------------------------------------------------- */

void Particle::add_tag()
{
	int maxtag = 0;
	for (int i = 0; i < nlocal; i++) 
		maxtag = MAX(maxtag,tag[i]);
	int maxtag_all;
	MPI_Allreduce(&maxtag,&maxtag_all,1,MPI_INT,MPI_MAX,mworld);

	// notag = # of particles with tag = 0 of one processor
	// notag_sum = total # of particles with tag = 0

	int notag = 0;
	for (int i = 0; i < nlocal; i++) {
		if (tag[i] == 0) {
			notag++;
		}
	}
	int notag_sum;
	MPI_Scan(&notag,&notag_sum,1,MPI_INT,MPI_SUM,mworld);

	// itag = 1st new tag that each processor should use

	int itag = maxtag_all + notag_sum - notag + 1;
	for (int i = 0; i < nlocal; i++) {
		if (tag[i] == 0) {
			tag[i] = itag;
			itag++;
		}
	}
}

/* ----------------------------------------------------------------------
   set a mass and flag it as set
   called from reading of data file
------------------------------------------------------------------------- */

void Particle::set_mass(const char *str)
{
	if (mass == NULL) error->all(FLERR,"Cannot set mass for this atom style");

	int itype;
	double mass_one;
	int n = sscanf(str,"%d %lg",&itype,&mass_one);
	if (n != 2) error->all(FLERR,"Invalid mass line in data file");

	if (itype < 1 || itype > ntypes)
		error->all(FLERR,"Invalid type for mass set");

	mass[itype] = mass_one;
	//mass_setflag[itype] = 1;

	if (mass[itype] <= 0.0) error->all(FLERR,"Invalid mass value");
}

/* ----------------------------------------------------------------------
   Set mass for each type of particle
------------------------------------------------------------------------- */

void Particle::set_mass(int narg, char** arg)
{
	int tid;

	// Need to tell if box exists
	if(mass == NULL) {
		allocate_type_arrays();
	}

	tid = atoi(arg[0]);                  // type id
	if (rmass_flag == 1) {
		double rm = atof(arg[1]);
		for (int i = 0; i < nlocal; i++) {
			if (type[i] == tid) rmass[i] = rm;
		}
	}
	else mass[tid] = atof(arg[1]);            // store mass

}

/* ----------------------------------------------------------------------
   Set density for each type of particle
------------------------------------------------------------------------- */

void Particle::set_density(int narg, char** arg)
{
	int tid;
	int gid = group->find_group(arg[0]);
	if (gid == -1) error->all(FLERR, "Cannot find the group id");

	int groupbit = group->bitmask[gid];

	if (sphere_flag == 0) error->all(FLERR, "Illegal particle style to call density command");

	if (!strcmp(arg[1], "set")) {
		if (narg != 3) error->all(FLERR, "Illegal radius command");
		for (int i = 0; i < nlocal; i++) {
			if (mask[i] & groupbit) {
				density[i] = atof(arg[2]);
			}
		}
	}


//	tid = atoi(arg[0]);
//	if (tid < 1) error->all(FLERR, "Illegal particle type");
	// Need to tell if box exists
//	if(density == NULL) {
//		allocate_type_arrays();
//	}

//	density[tid] = atof(arg[1]);
}

/* ----------------------------------------------------------------------
   Set radius for each type of particle
------------------------------------------------------------------------- */

void Particle::set_radius(int narg, char** arg)
{	
	if (sphere_flag == 0) error->all(FLERR, "Particle style is not correct to call radius command");
	if (narg < 1) error->all(FLERR, "Illegal radius command");

	int gid = group->find_group(arg[0]);
	if (gid == -1) error->all(FLERR, "Cannot find the group id");

	int groupbit = group->bitmask[gid];

	if (!strcmp(arg[1], "set")) {
		if (narg != 3) error->all(FLERR, "Illegal radius command");
		for (int i = 0; i < nlocal; i++) {
			if (mask[i] & groupbit) {
				radius[i] = atof(arg[2]);
				volume[i] = 4.0 / 3 * PI * radius[i] * radius[i] * radius[i];
				rmass[i] = density[i] * volume[i];
				
			}
		}
	}
	else if (!strcmp(arg[1], "create")) {
		if (narg != 9) error->all(FLERR, "Illegal radius command");
		/* The following needs to be changed to global gaussian distribution
		RanPark *random;
		int seed;
		double rlo, rhi, rmean, rsigma;
		rlo = atof(arg[2]);
		rhi = atof(arg[3]);
		rmean = atof(arg[4]);
		rsigma = atof(arg[5]);
		seed = atoi(arg[4]);
		random = new RanPark(ps, seed);
		double num;
		int count = 0;
		for (int i = 0; i < nlocal; i++) {
			num = rlo - 1;
			count = 0;
			while (num < rlo || num > rhi) {
				num = (random->gaussian())*rsigma + rmean;
				count++;
				if (count > 100000) {
					error->all(FLERR, "Cannot generate raidus for the required distribution");
				}
			}
			radius[i] = num;
		}
		*/
	}
	else error->all(FLERR, "Illegal raidus command");
}

/* ----------------------------------------------------------------------
Set energy for each type of particle
------------------------------------------------------------------------- */
void Particle::set_energy(int narg, char** arg)
{
	int tid;

//	if (atomic_flag == 1) error->all(FLERR, "Illegal particle style to call density command");
	if (narg != 2) error->all(FLERR, "Illegal density command");

//	tid = atoi(arg[0]);
//	if (tid < 1) error->all(FLERR, "Illegal particle type");
	// Need to tell if box exists
	int gid = group->find_group(arg[0]);
	if (gid == -1) error->all(FLERR, "Cannot find the group id");
	int groupbit = group->bitmask[gid];
	for (int i = 0; i < nlocal; i++) {
		if (mask[i] & groupbit) {
			e[i] = atof(arg[1]);
		}
	}
}

/* ----------------------------------------------------------------------
Set rho for each type of particle
------------------------------------------------------------------------- */
void Particle::set_rho(int narg, char** arg)
{
	int tid;
	//	if (atomic_flag == 1) error->all(FLERR, "Illegal particle style to call density command");
	if (narg != 2) error->all(FLERR, "Illegal density command");

	//	tid = atoi(arg[0]);
	//	if (tid < 1) error->all(FLERR, "Illegal particle type");
	// Need to tell if box exists
	int gid = group->find_group(arg[0]);
	if (gid == -1) error->all(FLERR, "Cannot find the group id");
	int groupbit = group->bitmask[gid];
	for (int i = 0; i < nlocal; i++) {
		if (mask[i] & groupbit) {
			rho[i] = atof(arg[1]);
		}
	}
}
/* ----------------------------------------------------------------------
   allocate arrays of length ntypes
   only done after ntypes is set
------------------------------------------------------------------------- */

void Particle::allocate_type_arrays()
{
	//if (avec->mass_type) {
	if (rmass_flag) {
		density = new double[ntypes+1];
	}
	else {
		mass = new double[ntypes+1];
	}
}

/* ----------------------------------------------------------------------
   unpack n lines from Atom section of data file
   call style-specific routine to parse line
------------------------------------------------------------------------- */

void Particle::data_particles(int n, char *buf)
{
	int m,imagedata,xptr,iptr;
	double xdata[3],lamda[3];
	double *coord;
	char *next;

	next = strchr(buf,'\n');
	*next = '\0';
	int nwords = count_words(buf);
	*next = '\n';

	if (nwords != size_data_atom && nwords != size_data_atom + 3) {
		error->all(FLERR,"Incorrect atom format in data file");
	}

	char **values = new char*[nwords];

	// set bounds for my proc
	// if periodic and I am lo/hi proc, adjust bounds by EPSILON
	// insures all data atoms will be owned even with round-off

	double epsilon[3];

    epsilon[0] = domain->boxle[0] * EPSILON;
    epsilon[1] = domain->boxle[1] * EPSILON;
    epsilon[2] = domain->boxle[2] * EPSILON;
  
	double sublo[3],subhi[3];
 
    sublo[0] = domain->sublo[0]; subhi[0] = domain->subhi[0];
    sublo[1] = domain->sublo[1]; subhi[1] = domain->subhi[1];
    sublo[2] = domain->sublo[2]; subhi[2] = domain->subhi[2];
  
	if (domain->xperiodic) {
		if (parallel->procloc[0] == 0) sublo[0] -= epsilon[0];
		if (parallel->procloc[0] == parallel->procgrid[0]-1) subhi[0] += epsilon[0];
	}
	if (domain->yperiodic) {
		if (parallel->procloc[1] == 0) sublo[1] -= epsilon[1];
		if (parallel->procloc[1] == parallel->procgrid[1]-1) subhi[1] += epsilon[1];
	}
	if (domain->zperiodic) {
		if (parallel->procloc[2] == 0) sublo[2] -= epsilon[2];
		if (parallel->procloc[2] == parallel->procgrid[2]-1) subhi[2] += epsilon[2];
	}

	// xptr = which word in line starts xyz coords
	// iptr = which word in line starts ix,iy,iz image flags

	xptr = xcol_data - 1;

	// loop over lines of atom data
	// tokenize the line into values
	// extract xyz coords and image flags
	// remap atom into simulation box
	// if atom is in my sub-domain, unpack its values

	for (int i = 0; i < n; i++) {
		next = strchr(buf,'\n');

		values[0] = strtok(buf," \t\n\r\f");
		if (values[0] == NULL)
		  error->all(FLERR,"Incorrect atom format in data file");
		for (m = 1; m < nwords; m++) {
		  values[m] = strtok(NULL," \t\n\r\f");
		  if (values[m] == NULL)
			error->all(FLERR,"Incorrect atom format in data file");
		}

		xdata[0] = atof(values[xptr]);
		xdata[1] = atof(values[xptr+1]);
		xdata[2] = atof(values[xptr+2]);
		//domain->remap(xdata,imagedata);
		coord = xdata;

		if (coord[0] >= sublo[0] && coord[0] < subhi[0] &&
			coord[1] >= sublo[1] && coord[1] < subhi[1] &&
			coord[2] >= sublo[2] && coord[2] < subhi[2])
			ptype->data_particle(xdata,values);

		buf = next + 1;
	}

	delete [] values;
}

/* ----------------------------------------------------------------------
   unpack n lines from Velocity section of data file
   check that atom IDs are > 0 and <= map_tag_max
   call style-specific routine to parse line
------------------------------------------------------------------------- */

void Particle::data_vels(int n, char *buf)
{
	int j,m,tagdata;
	char *next;

	next = strchr(buf,'\n');
	*next = '\0';
	int nwords = count_words(buf);
	*next = '\n';

	if (nwords != size_data_vel)
		error->all(FLERR,"Incorrect velocity format in data file");

	char **values = new char*[nwords];

	// loop over lines of atom velocities
	// tokenize the line into values
	// if I own atom tag, unpack its values

	for (int i = 0; i < n; i++) {
		next = strchr(buf,'\n');

		values[0] = strtok(buf," \t\n\r\f");
		for (j = 1; j < nwords; j++)
			values[j] = strtok(NULL," \t\n\r\f");

		tagdata = atoi(values[0]);
		if (tagdata <= 0 || tagdata > map_tag_max)
			error->one(FLERR,"Invalid atom ID in Velocities section of data file");
		if ((m = map(tagdata)) >= 0) {
			ptype->data_vel(m,&values[1]);
		}

		buf = next + 1;
	}

	delete [] values;
}

/* ----------------------------------------------------------------------
   count and return words in a single line
   make copy of line before using strtok so as not to change line
   trim anything from '#' onward
------------------------------------------------------------------------- */

int Particle::count_words(const char *line)
{
	int n = strlen(line) + 1;
	char *copy;
	memory->create(copy,n,"atom:copy");
	strcpy(copy,line);

	char *ptr;
	if (ptr = strchr(copy,'#')) *ptr = '\0';

	if (strtok(copy," \t\n\r\f") == NULL) {
		memory->destroy(copy);
		return 0;
	}
	n = 1;
	while (strtok(NULL," \t\n\r\f")) n++;

	memory->destroy(copy);
	return n;
}

/* ----------------------------------------------------------------------
   Allocate and initialize array table for global -> local map
   set map_tag_max = largest atom ID (may be larger than natoms)
   for array option:
     array length = 1 to largest tag of any atom
     set entire array to -1 as initial values
------------------------------------------------------------------------- */

void Particle::map_init()
{
	map_delete();

	if (tag_enable == 0) {
		error->all(FLERR,"Cannot create a particle map unless particles have IDs");
	}

	int max = 0;
	for (int i = 0; i < nlocal; i++) max = MAX(max,tag[i]);
	MPI_Allreduce(&max,&map_tag_max,1,MPI_INT,MPI_MAX,mworld);

	if (map_style == 1) {
		memory->create(map_array,map_tag_max+1,"atom:map_array");
		for (int i = 0; i <= map_tag_max; i++) map_array[i] = -1;
	}
	else {
		// map_nhash = max of atoms/proc or total atoms, times 2, at least 1000

		int nper = static_cast<int> (nparticles/parallel->nprocs);
		map_nhash = MAX(nper, nmax);
		if (map_nhash > nparticles) map_nhash = static_cast<int> (nparticles);
		if (parallel->nprocs > 1) map_nhash *= 2;
		map_nhash = MAX(map_nhash,1000);

		// map_nbucket = prime just larger than map_nhash

		int n = map_nhash/10000;
		n = MIN(n, nprimes-1);
		map_nbucket = primes[n];
		if (map_nbucket < map_nhash && n < nprimes-1) map_nbucket = primes[n+1];

		// set all buckets to empty
		// set hash to map_nhash in length
		// put all hash entries in free list and point them to each other

		map_bucket = new int[map_nbucket];
		for (int i = 0; i < map_nbucket; i++) map_bucket[i] = -1;

		map_hash = new HashElem[map_nhash];
		map_nused = 0;
		map_free = 0;
		for (int i = 0; i < map_nhash; i++) map_hash[i].next = i+1;
		map_hash[map_nhash-1].next = -1;
	}
}

/* ----------------------------------------------------------------------
   Clear global -> local map for all of my own and ghost atoms
------------------------------------------------------------------------- */

void Particle::map_clear()
{
	if (map_style == 1) {
		int nall = nlocal + nghost;
		for (int i = 0; i < nall; i++) map_array[tag[i]] = -1;
	} 
	else {
		int previous,global,ibucket,index;
		int nall = nlocal + nghost;
		for (int i = 0; i < nall; i++) {
			// search for key
			// if don't find it, done

			previous = -1;
			global = tag[i];
			ibucket = global % map_nbucket;
			index = map_bucket[ibucket];
			while (index > -1) {
				if (map_hash[index].global == global) break;
				previous = index;
				index = map_hash[index].next;
			}
			if (index == -1) continue;

			// delete the hash entry and add it to free list
			// special logic if entry is 1st in the bucket

			if (previous == -1) map_bucket[ibucket] = map_hash[index].next;
			else map_hash[previous].next = map_hash[index].next;

			map_hash[index].next = map_free;
			map_free = index;
			map_nused--;
		}
	}
	
}

/* ----------------------------------------------------------------------
   set global -> local map for all of my own and ghost atoms
   loop in reverse order so that nearby images take precedence over far ones
     and owned atoms take precedence over images
   this enables valid lookups of bond topology atoms
------------------------------------------------------------------------- */

void Particle::map_set()
{
	if (map_style == 1) {
		int nall = nlocal + nghost;
		for (int i = nall-1; i >= 0 ; i--) map_array[tag[i]] = i;
	} 
	else {
		int previous,global,ibucket,index;
		int nall = nlocal + nghost;
		if (nall > map_nhash) map_init();

		for (int i = nall-1; i >= 0 ; i--) {
		    // search for key
			// if found it, just overwrite local value with index

			previous = -1; 
			global = tag[i];
			ibucket = global % map_nbucket;
			index = map_bucket[ibucket];
			while (index > -1) {
				if (map_hash[index].global == global) break;
				previous = index;
				index = map_hash[index].next;
			}
			if (index > -1) {
				map_hash[index].local = i;
				continue;
			}

			// take one entry from free list
			// add the new global/local pair as entry at end of bucket list
			// special logic if this entry is 1st in bucket

			index = map_free;
			map_free = map_hash[map_free].next;
			if (previous == -1) map_bucket[ibucket] = index;
			else map_hash[previous].next = index;
			map_hash[index].global = global;
			map_hash[index].local = i;
			map_hash[index].next = -1;
			map_nused++;
		}
	}
}

/* ----------------------------------------------------------------------
   set global to local map for one atom
   for hash table option:
     global ID may already be in table if atom was already set
------------------------------------------------------------------------- */

void Particle::map_one(int global, int local)
{
	if (map_style == 1) map_array[global] = local;
	else {
		// search for key
		// if found it, just overwrite local value with index

		int previous = -1;
		int ibucket = global % map_nbucket;
		int index = map_bucket[ibucket];
		while (index > -1) {
			if (map_hash[index].global == global) break;
			previous = index;
			index = map_hash[index].next;
		}
		if (index > -1) {
			map_hash[index].local = local;
			return;
		}

		// take one entry from free list
		// add the new global/local pair as entry at end of bucket list
		// special logic if this entry is 1st in bucket

		index = map_free;
		map_free = map_hash[map_free].next;
		if (previous == -1) map_bucket[ibucket] = index;
		else map_hash[previous].next = index;
		map_hash[index].global = global;
		map_hash[index].local = local;
		map_hash[index].next = -1;
		map_nused++;
	}
}

/* ----------------------------------------------------------------------
   Free the array table for global to local mapping
------------------------------------------------------------------------- */

void Particle::map_delete()
{
	if (map_style == 1) {
		if (map_tag_max) memory->destroy(map_array);
	}
	else {
		if (map_nhash) {
			delete [] map_bucket;
			delete [] map_hash;
		}
		map_nhash = 0;
	}
	
	map_tag_max = 0;
}

/* ----------------------------------------------------------------------
   lookup global ID in hash table, return local index
------------------------------------------------------------------------- */

int Particle::map_find_hash(int global)
{
	int local = -1;
	int index = map_bucket[global % map_nbucket];
	while (index > -1) {
		if (map_hash[index].global == global) {
			local = map_hash[index].local;
			break;
		}
		index = map_hash[index].next;
	}
	return local;
}

/* ----------------------------------------------------------------------
   Set position for each particle
------------------------------------------------------------------------- */

void Particle::set_pos(int narg, char** arg)
{                


}

/* ---------------------------------------------------------------------- */

void Particle::lost_check()
{
	int ntotal;

	MPI_Allreduce(&nlocal, &ntotal, 1, MPI_INT, MPI_SUM, mworld);

	if (ntotal < nparticles) {
		char str[128];
		sprintf(str, "Particle lost from %d total particles to %d", particle->nparticles, ntotal);
		error->warning(FLERR, str);
	}
}

/* ----------------------------------------------------------------------
Delete particle
------------------------------------------------------------------------- */

void Particle::delete_particle(int n)
{
	int nlocal = particle->nlocal;
	int i;
	if (n < nlocal){
		tag[n] = tag[nlocal];
		type[n] = type[nlocal]; 
		mask[n] = mask[nlocal];
		for (i = 0; i < 3; i++) {
			x[n][i] = x[nlocal][i];
			v[n][i] = v[nlocal][i];
			f[n][i] = f[nlocal][i];
		
		}
	}
	if (atomic_flag == 1) 
		mass[n] = mass[nlocal];
	else {
		for (i = 0; i < 3; i++){
			omega[n][i] = omega[nlocal][i];
			torque[n][i] = torque[nlocal][i];
		}
		radius[n] = radius[nlocal];
		rmass[n] = rmass[nlocal];
		poro[n] = poro[nlocal];
		volume[n] = volume[nlocal];
		hlocal[n] = volume[nlocal];
	}

	particle->nlocal--;
}

/* ----------------------------------------------------------------------
Save particle
------------------------------------------------------------------------- */

void Particle::save_particle(int narg, char** arg)
{
	if (!strcmp(arg[0], "cylinder")) {
		if (narg != 6) error->all(FLERR, "Illegal save command");
		double coord[3], height, radius;
		coord[0] = atof(arg[1]);
		coord[1] = atof(arg[2]);
		coord[2] = atof(arg[3]);
		height = atof(arg[4]);
		radius = atof(arg[5]);
		int nlocal = particle->nlocal;
		int i;
		double rijsq;
		for (i = nlocal; i >= 0; i--) {
			rijsq = (x[i][0] - coord[0]) * (x[i][0] - coord[0]) + (x[i][1] - coord[1]) * (x[i][1] - coord[1]);
			if(rijsq < radius * radius && x[i][2] > coord[2] && x[i][2] < coord[2] + height)
				continue;
			else
				delete_particle(i);
		}
	}
}

//	transfer data to easy fetch memory
/* ---------------------------------------------------------------------- */
void Particle::PinHostArray(){
	hipDeviceSynchronize();
	if (nmax){
			ptrHostCoord = cudaEngine->PinHost(devHostCoord, &(particle->x[0][0]), 3 * particle->nmax * sizeof(double));
			ptrHostForce = cudaEngine->PinHost(devHostForce, &(particle->f[0][0]), 3 * particle->nmax * sizeof(double));
			ptrHostVelo = cudaEngine->PinHost(devHostVelo, &(particle->v[0][0]), 3 * particle->nmax * sizeof(double));
			ptrHostVest = cudaEngine->PinHost(devHostVest, &(particle->vest[0][0]), 3 * particle->nmax * sizeof(double));
			ptrHostMassType = cudaEngine->PinHost(devHostMassType, particle->mass, (particle->ntypes + 1) * sizeof(double));
			ptrHostType = cudaEngine->PinHost(devHostType, particle->type, particle->nmax * sizeof(int));
			ptrHostTag = cudaEngine->PinHost(devHostTag, particle->tag, particle->nmax * sizeof(int));
			ptrHostMask = cudaEngine->PinHost(devHostMask, particle->mask, particle->nmax * sizeof(int));
	}
	if (rho) ptrHostRho = cudaEngine->PinHost(devHostRho, particle->rho, particle->nmax * sizeof(double));
	if (rmass)	ptrHostRmass = cudaEngine->PinHost(devHostRmass, particle->rmass, particle->nmax * sizeof(double));
	if (density) ptrHostDensity = cudaEngine->PinHost(devHostDensity, particle->density, particle->nmax * sizeof(double));
	if (radius) ptrHostRadius = cudaEngine->PinHost(devHostRadius, particle->radius, particle->nmax * sizeof(double));
	if (poro) ptrHostPoro = cudaEngine->PinHost(devHostPoro, particle->poro, particle->nmax * sizeof(double));
	if (volume) ptrHostVolume = cudaEngine->PinHost(devHostVolume, particle->volume, particle->nmax * sizeof(double));

}

/* ---------------------------------------------------------------------- */
void Particle::UnpinHostArray(){
	hipDeviceSynchronize();
	if (ptrHostCoord)    cudaEngine->UnpinHost(&(particle->x[0][0]), ptrHostCoord, devHostCoord);
	if (ptrHostForce)    cudaEngine->UnpinHost(&(particle->f[0][0]), ptrHostForce, devHostForce);
	if (ptrHostVelo)     cudaEngine->UnpinHost(&(particle->v[0][0]), ptrHostVelo, devHostVelo);
	if (ptrHostVest)     cudaEngine->UnpinHost(&(particle->vest[0][0]), ptrHostVest, devHostVest);
	if (ptrHostRho)    cudaEngine->UnpinHost(particle->rho, ptrHostRho, devHostRho);
	if (ptrHostType)    cudaEngine->UnpinHost(particle->type, ptrHostType, devHostType);
	if (ptrHostTag)    cudaEngine->UnpinHost(particle->tag, ptrHostTag, devHostTag);
	if (ptrHostMask)    cudaEngine->UnpinHost(particle->mask, ptrHostMask, devHostMask);
	if (ptrHostMassType) cudaEngine->UnpinHost(particle->mass, ptrHostMassType, devHostMassType);
	if (ptrHostRmass) cudaEngine->UnpinHost(particle->rmass, ptrHostRmass, devHostRmass);
	if (ptrHostRadius) cudaEngine->UnpinHost(particle->radius, ptrHostRadius, devHostRadius);
	if (ptrHostDensity) cudaEngine->UnpinHost(particle->density, ptrHostDensity, devHostDensity);
	if (ptrHostPoro) cudaEngine->UnpinHost(particle->poro, ptrHostPoro, devHostPoro);
	if (ptrHostVolume) cudaEngine->UnpinHost(particle->volume, ptrHostVolume, devHostVolume);

}

//	transfer data From CPU to GPU
/* ---------------------------------------------------------------------- */
void Particle::TransferC2G(){

	enum hipLimit_t 	limit;
	size_t *memory, *free, *total;
	memory = (size_t *)malloc(sizeof(size_t));
	free = (size_t *)malloc(sizeof(size_t));
	total = (size_t *)malloc(sizeof(size_t));
	int *device_num;
	device_num = (int *)malloc(sizeof(int));
	hipError_t cudaStatus = hipGetDevice(device_num);
	cudaStatus = hipMemGetInfo(free, total);
	//cudaStatus = hipDeviceGetLimit(memory, limit);



	int nlocal = particle->nlocal;
	if (nlocal == 0)
		return;

	const int BLK = 512;
	//	Therr are maximum 16 stream to run the code transfer concurrently
	int nStream = cudaEngine->StreamPool.size();
	vector<hipStream_t> StreamPool = cudaEngine->StreamPool;
	vector<hipEvent_t> Events;

	//	Transfer each terms on different streams and record the event
	hipStream_t &Stream = StreamPool[0];
	hipMemcpyAsync(devHostCoord, ptrHostCoord, 3 * nlocal * sizeof(double), hipMemcpyHostToDevice, Stream);
	gpuDeinterLeave << < int(nlocal + BLK - 1) / BLK, BLK, BLK * 3 * sizeof(double), Stream >> >(
		devHostCoord, devCoordX, devCoordY, devCoordZ, nlocal);
	Events.push_back(cudaEngine->Event("X_C2G_PREV"));
	hipEventRecord(Events.back(), Stream);

	Stream = StreamPool[1];
	hipMemcpyAsync(devHostVelo, ptrHostVelo, 3 * nlocal * sizeof(double), hipMemcpyHostToDevice, Stream);
	gpuDeinterLeave << < int(nlocal + BLK - 1) / BLK, BLK, BLK * 3 * sizeof(double), Stream >> >(
		devHostVelo, devVeloX, devVeloY, devVeloZ, nlocal);
	Events.push_back(cudaEngine->Event("V_C2G_PREV"));
	hipEventRecord(Events.back(), Stream);

	Stream = StreamPool[2];
	hipMemcpyAsync(devHostForce, ptrHostForce, 3 * nlocal * sizeof(double), hipMemcpyHostToDevice, Stream);
	gpuDeinterLeave << < int(nlocal + BLK - 1) / BLK, BLK, BLK * 3 * sizeof(double), Stream >> >(
		devHostForce, devForceX, devForceY, devForceZ, nlocal);
	Events.push_back(cudaEngine->Event("F_C2G_PREV"));
	hipEventRecord(Events.back(), Stream);

	Stream = StreamPool[3];
	hipMemcpyAsync(devHostType, ptrHostType, nlocal * sizeof(int), hipMemcpyHostToDevice, Stream);
	hipMemcpyAsync(devHostMask, ptrHostMask, nlocal * sizeof(int), hipMemcpyHostToDevice, Stream);
	hipMemcpyAsync(devHostTag, ptrHostTag, nlocal * sizeof(int), hipMemcpyHostToDevice, Stream);
	hipMemcpyAsync(devHostMassType, ptrHostMassType, (ntypes + 1) * sizeof(double), hipMemcpyHostToDevice, Stream);
	gpuCopy << < int(nlocal + BLK - 1) / BLK, BLK, 0, Stream >> > (devType, devHostType, nlocal);
	gpuCopy << < int(nlocal + BLK - 1) / BLK, BLK, 0, Stream >> > (devMask, devHostMask, nlocal);
	gpuCopy << < int(nlocal + BLK - 1) / BLK, BLK, 0, Stream >> > (devTag, devHostTag, nlocal);
	gpuCopy << < int(nlocal + BLK - 1) / BLK, BLK, 0, Stream >> > (devMass, devHostMassType, ntypes + 1);
	Events.push_back(cudaEngine->Event("T_C2G_PREV"));
	hipEventRecord(Events.back(), Stream);

	Stream = StreamPool[4];
	hipMemcpyAsync(devHostRadius, ptrHostRadius, nlocal * sizeof(double), hipMemcpyHostToDevice, Stream);
	hipMemcpyAsync(devHostRmass, ptrHostRmass, nlocal * sizeof(double), hipMemcpyHostToDevice, Stream);
	hipMemcpyAsync(devHostRho, ptrHostRho, nlocal * sizeof(double), hipMemcpyHostToDevice, Stream);
	gpuCopy << < int(nlocal + BLK - 1) / BLK, BLK, 0, Stream >> > (devRadius, devHostRadius, nlocal);
	gpuCopy << < int(nlocal + BLK - 1) / BLK, BLK, 0, Stream >> > (devRmass, devHostRmass, nlocal);
	gpuCopy << < int(nlocal + BLK - 1) / BLK, BLK, 0, Stream >> > (devRho, devHostRho, nlocal);
	Events.push_back(cudaEngine->Event("R_C2G_PREV"));
	hipEventRecord(Events.back(), Stream);

	Stream = StreamPool[5];
	hipMemcpyAsync(devHostDensity, ptrHostDensity, nlocal * sizeof(double), hipMemcpyHostToDevice, Stream);
	hipMemcpyAsync(devHostPoro, ptrHostPoro, nlocal * sizeof(double), hipMemcpyHostToDevice, Stream);
	hipMemcpyAsync(devHostVolume, ptrHostVolume, nlocal * sizeof(double), hipMemcpyHostToDevice, Stream);
	gpuCopy << < int(nlocal + BLK - 1) / BLK, BLK, 0, Stream >> > (devDensity, devHostDensity, nlocal);
	gpuCopy << < int(nlocal + BLK - 1) / BLK, BLK, 0, Stream >> > (devPoro, devHostPoro, nlocal);
	gpuCopy << < int(nlocal + BLK - 1) / BLK, BLK, 0, Stream >> > (devVolume, devHostVolume, nlocal);
	Events.push_back(cudaEngine->Event("D_C2G_PREV"));
	hipEventRecord(Events.back(), Stream);

	Stream = StreamPool[6];
	hipMemcpyAsync(devHostVest, ptrHostVest, 3 * nlocal * sizeof(double), hipMemcpyHostToDevice, Stream);
	gpuDeinterLeave << < int(nlocal + BLK - 1) / BLK, BLK, BLK * 3 * sizeof(double), Stream >> >(
		devHostVest, devVestX, devVestY, devVestZ, nlocal);
	Events.push_back(cudaEngine->Event("V_C2G_PREV"));
	hipEventRecord(Events.back(), Stream);

	//	Make sure they all stop before end the function
	for (int i = 0; i < Events.size(); i++)
		hipStreamWaitEvent(StreamPool[0], Events[i], 0);

}

__global__ void add(int *a, int *b, int *c) {
	c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
}

//	transfer data From GPU to CPU
/* ---------------------------------------------------------------------- */
void Particle::TransferG2C(){

	//enum hipLimit_t 	limit;
	//size_t *memory, *free, *total;
	//memory = (size_t *)malloc(sizeof(size_t));
	//free = (size_t *)malloc(sizeof(size_t));
	//total = (size_t *)malloc(sizeof(size_t));
	//int *device_num;
	//device_num = (int *)malloc(sizeof(int));
	//hipError_t cudaStatus = hipGetDevice(device_num);
	//cudaStatus = hipMemGetInfo(free, total);
	////cudaStatus = hipDeviceGetLimit(memory, limit);
	hipEvent_t start, stop;
	float time;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	int nlocal = particle->nlocal;
	if (nlocal == 0)
		return;

	const int BLK = 512;

	//	Therr are maximum 16 stream to run the code transfer concurrently
	int nStream = cudaEngine->StreamPool.size();
	vector<hipStream_t> StreamPool = cudaEngine->StreamPool;
	vector<hipEvent_t> Events;

	//	Transfer each terms on different streams and record the event
	hipStream_t &Stream = StreamPool[0];
	gpuInterLeave << < int(nlocal + BLK - 1) / BLK, BLK, BLK * 3 * sizeof(double), Stream >> >(
		devHostCoord, devCoordX, devCoordY, devCoordZ, nlocal); 
	hipMemcpyAsync(ptrHostCoord, devHostCoord, 3 * nlocal * sizeof(double), hipMemcpyDeviceToHost, Stream);
	Events.push_back(cudaEngine->Event("X_G2C_PREV"));
	hipEventRecord(Events.back(), Stream);

	Stream = StreamPool[1];
	gpuInterLeave << < int(nlocal + BLK - 1) / BLK, BLK, BLK * 3 * sizeof(double), Stream >> >(
		devHostVelo, devVeloX, devVeloY, devVeloZ, nlocal);
	hipMemcpyAsync(ptrHostVelo, devHostVelo, 3 * nlocal * sizeof(double), hipMemcpyDeviceToHost, Stream);
	Events.push_back(cudaEngine->Event("V_G2C_PREV"));
	hipEventRecord(Events.back(), Stream);

	Stream = StreamPool[2];
	gpuInterLeave << < int(nlocal + BLK - 1) / BLK, BLK, BLK * 3 * sizeof(double), Stream >> >(
		devHostForce, devForceX, devForceY, devForceZ, nlocal);
	hipMemcpyAsync(ptrHostForce, devHostForce, 3 * nlocal * sizeof(double), hipMemcpyDeviceToHost, Stream);
	Events.push_back(cudaEngine->Event("F_G2C_PREV"));
	hipEventRecord(Events.back(), Stream);

	Stream = StreamPool[3];
	hipMemcpyAsync(ptrHostType, devType, nlocal * sizeof(int), hipMemcpyDeviceToHost, Stream);
	hipMemcpyAsync(ptrHostMask, devMask, nlocal * sizeof(int), hipMemcpyDeviceToHost, Stream);
	hipMemcpyAsync(ptrHostTag, devTag, nlocal * sizeof(int), hipMemcpyDeviceToHost, Stream);
	hipMemcpyAsync(ptrHostMassType, devMass, nlocal * sizeof(double), hipMemcpyDeviceToHost, Stream);
	Events.push_back(cudaEngine->Event("T_G2C_PREV"));
	hipEventRecord(Events.back(), Stream);

	Stream = StreamPool[4];
	gpuCopy << < int(nlocal + BLK - 1) / BLK, BLK, 0, Stream >> > (devRadius, devHostRadius, nlocal);
	gpuCopy << < int(nlocal + BLK - 1) / BLK, BLK, 0, Stream >> > (devRmass, devHostRmass, nlocal);
	hipMemcpyAsync(ptrHostRadius, devRadius, nlocal * sizeof(double), hipMemcpyDeviceToHost, Stream);
	hipMemcpyAsync(ptrHostRmass, devRmass, nlocal * sizeof(double), hipMemcpyDeviceToHost, Stream);
	hipMemcpyAsync(ptrHostRho, devRho, nlocal * sizeof(double), hipMemcpyDeviceToHost, Stream);
	Events.push_back(cudaEngine->Event("R_G2C_PREV"));
	hipEventRecord(Events.back(), Stream);

	Stream = StreamPool[5];
	hipMemcpyAsync(ptrHostDensity, devDensity, nlocal * sizeof(double), hipMemcpyDeviceToHost, Stream);
	hipMemcpyAsync(ptrHostPoro, devPoro, nlocal * sizeof(double), hipMemcpyDeviceToHost, Stream);
	hipMemcpyAsync(ptrHostVolume, devVolume, nlocal * sizeof(double), hipMemcpyDeviceToHost, Stream);
	Events.push_back(cudaEngine->Event("D_G2C_PREV"));
	hipEventRecord(Events.back(), Stream);

	Stream = StreamPool[6];
	gpuInterLeave << < int(nlocal + BLK - 1) / BLK, BLK, BLK * 3 * sizeof(double), Stream >> >(
		devHostVest, devVestX, devVestY, devVestZ, nlocal);
	hipMemcpyAsync(ptrHostVest, devHostVest, 3 * nlocal * sizeof(double), hipMemcpyDeviceToHost, Stream);
	Events.push_back(cudaEngine->Event("V_G2C_PREV"));
	hipEventRecord(Events.back(), Stream);

	//	Make sure they all stop before end the function
	for (int i = 0; i < Events.size(); i++)
		hipStreamWaitEvent(StreamPool[0], Events[i], 0);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	time = time;
}